#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *arr, int n) {
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        arr[i] = i;
    }
}

int main() {
    int n = 114;
    int *arr;
    hipMallocManaged(&arr, n * sizeof(int));

    kernel<<<1, 4>>>(arr, n);

    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < n; i++) {
        printf("arr[%d]: %d\n", i, arr[i]);
    }

    hipFree(arr);
    return 0;
}

