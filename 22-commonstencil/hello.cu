#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/universal_vector.h>
#include <chrono>

using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::seconds;
using std::chrono::system_clock;

__global__ void stencil(int row_num, int col_num, int *arr_data, int *result) {
    auto index = blockIdx.x * blockDim.x + threadIdx.x;
    auto current_row = index / col_num;
    auto current_col = index % col_num;
    auto data0 = arr_data[index];
    // up
    auto data1 = arr_data[(current_row + row_num - 1) % row_num * col_num + current_col];
    // down
    auto data2 = arr_data[(current_row + 1) % row_num * col_num + current_col];
    // left
    auto data3 = arr_data[current_row * col_num + (current_col + col_num - 1) % col_num ];
    // right
    auto data4 = arr_data[current_row * col_num + (current_col + 1) % col_num];

    result[index] = data1 + data2 + data3 + data4 - 4 * data0;
}

int main() {
    int row_num = 1 << 14;
    int col_num = 1 << 14;

    int *arr;
    int *result;
    hipMallocManaged(&arr, sizeof(int) * row_num * col_num);
    hipMallocManaged(&result, sizeof(int) * row_num * col_num);

    for (int index = 0; index < row_num * col_num; ++index) {
        arr[index] = rand() % 1024 - 512;
    }

    hipDeviceSynchronize();
    auto begin_millis = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();

    hipDeviceSynchronize();
    int total_numbers = row_num * col_num;
    int block_size = 1024;
    stencil<<<total_numbers / block_size, block_size>>>(row_num, col_num, arr, result);

    hipDeviceSynchronize(); 
    auto end_millis = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
    printf("%ld\n", end_millis - begin_millis);
    hipDeviceSynchronize();
    return 0;
}

