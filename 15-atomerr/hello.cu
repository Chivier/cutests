#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__device__ float sum = 0;

template <class Func>
__global__ void kernel(int n, Func func) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x) {
        func(i);
    }
}

int main() {
    int n = 65536;
    int *arr;
    float result = 0;

    hipMallocManaged(&arr, n * sizeof(int));

    int block_dim = 128;
    int grid_dim = (n - 1) / block_dim;
    kernel<<<grid_dim, block_dim>>>(n, [=] __device__ (int i) {
        arr[i] = i;
    });
    
    
    kernel<<<grid_dim, block_dim>>>(n, [=] __device__ (int i) {
        sum += sinf(arr[i]);
    });

    hipMemcpyFromSymbol(&result, HIP_SYMBOL(sum), sizeof(float), 0, hipMemcpyDeviceToHost);
    checkCudaErrors(hipDeviceSynchronize());
    
    printf("%f\n", result);

    // Compare
    result = 0;
    for(int index = 0; index < n; ++index) {
        result += sinf(index);
    }
    printf("%f", result);

    hipFree(arr);
    return 0;
}
