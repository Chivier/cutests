#include <stdio.h>
#include <hip/hip_runtime.h>

float p2p_copy (size_t size) {
    int gpu_numbers = 100;
    int *pointers[gpu_numbers];
    
    for (int index = 0; index < gpu_numbers; ++index) {
        hipSetDevice(index);
        hipMalloc(&pointers[index], size);
    }

    for (int indexi = 0; indexi < gpu_numbers; ++indexi) {
        hipSetDevice(indexi);
        for (int indexj = 0; indexj < gpu_numbers; ++indexj) {
            if (indexi == indexj)
                continue;
            hipDeviceEnablePeerAccess(indexj, 0);
        }
    }
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
  
    hipEventRecord(begin);
    for (int TTT = 100; TTT >= 0; --TTT)
    for (int index = 1; index < gpu_numbers; ++index) {
        hipMemcpyAsync(pointers[0], pointers[index], size, hipMemcpyDeviceToDevice);
    }
    hipEventRecord(end);
    hipEventSynchronize (end);
  
    float elapsed;
    hipEventElapsedTime (&elapsed, begin, end);
    elapsed /= 1000;
  
    hipSetDevice (0);
    hipFree (pointers[0]);
  
    hipSetDevice (1);
    hipFree (pointers[1]);
  
    hipEventDestroy (end);
    hipEventDestroy (begin);
  
    return elapsed;
}

int main() {
    auto time = p2p_copy(1000000000);
    printf("time = %f s\n", time);
    return 0;
}

