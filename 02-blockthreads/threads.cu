#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void kernel() {
    printf("Block %d of %d, Thread %d of %d\n",
           blockIdx.x, gridDim.x, threadIdx.x, blockDim.x);
}

int main() {
    kernel<<<4, 3>>>();
    hipDeviceSynchronize();
    return 0;
}

