#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *arr) {
    arr[0] = 0;
    int index = 1;
    while (arr[index] != 0) {
        arr[0] += arr[index];
        index++;
    }
}

int main() {
    int *a;
    a = (int *)malloc(sizeof(int) * 12);
    int index = 1;
    for (index = 1; index <= 10; ++index) {
        a[index] = index;
    }

    int *cuda_a;
    hipMalloc(&cuda_a, sizeof(int) * 12);
    hipMemcpy(cuda_a, a, sizeof(int) * 12, hipMemcpyHostToDevice);
    kernel<<<1, 1>>>(cuda_a);
    hipMemcpy(a, cuda_a, sizeof(int) * 12, hipMemcpyDeviceToHost);

    checkCudaErrors(hipDeviceSynchronize());
    printf("%d\n", a[0]);
    free(a);
    hipFree(cuda_a);
    return 0;
}

