#include "hip/hip_runtime.h"
#include <cstdio>  
#include <hip/hip_runtime.h>  
#include "hip/hip_runtime_api.h"  
  
template <class Func>  
__global__ void kernel(int n, Func func) {  
   for (int i = blockDim.x * blockIdx.x + threadIdx.x;  
        i < n; i += blockDim.x * gridDim.x) {  
       func(i);  
   }  
}  
  
int main() {  
   int n = 10;  
   int *arr;  
  
   hipMallocManaged(&arr, n * sizeof(int));  
  
   int block_dim = 128;  
   int grid_dim = (n - 1) / block_dim + 1;  
   kernel<<<grid_dim, block_dim>>>(n, [=] __device__ (int i) {  
       arr[i] = i;  
   });  
      
   checkCudaErrors(hipDeviceSynchronize());  
   kernel<<<grid_dim, block_dim>>>(n, [=] __device__ (int i) {  
       printf("%d, %f\n", i, sinf(arr[i]));  
   });  
  
   checkCudaErrors(hipDeviceSynchronize());  
  
   // Compare  
   // for(int index = 0; index < n; ++index) {  
   //    printf("%d, %f\n", index, sinf(index));  
   //}  
  
   hipFree(arr);  
   return 0;  
}
