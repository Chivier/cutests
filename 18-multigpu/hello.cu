#include <iostream>
#include <hip/hip_runtime.h>

float p2p_copy (size_t size) {
    int *pointers[2];
  
    hipSetDevice (0);
    hipDeviceEnablePeerAccess (1, 0);
    hipMalloc (&pointers[0], size);
  
    hipSetDevice (1);
    hipDeviceEnablePeerAccess (0, 0);
    hipMalloc (&pointers[1], size);
  
    hipEvent_t begin, end;
    hipEventCreate (&begin);
    hipEventCreate (&end);
  
    hipEventRecord (begin);
    hipMemcpyAsync (pointers[0], pointers[1], size, hipMemcpyDeviceToDevice);
    hipEventRecord (end);
    hipEventSynchronize (end);
  
    float elapsed;
    hipEventElapsedTime (&elapsed, begin, end);
    elapsed /= 1000;
  
    hipSetDevice (0);
    hipFree (pointers[0]);
  
    hipSetDevice (1);
    hipFree (pointers[1]);
  
    hipEventDestroy (end);
    hipEventDestroy (begin);
  
    return elapsed;
}

int main() {
    auto time = p2p_copy(1000000);
    printf("time = %f\n", time);
    return 0;
}

